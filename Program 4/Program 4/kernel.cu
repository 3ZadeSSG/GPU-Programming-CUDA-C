#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>
#include<math.h>
#include<iomanip>
#define dd double
#define n 4
using namespace std;
__global__ void euclidianDistance(dd *p1, dd *p2, dd *result,dd *temp) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	*result = 0;
	temp[i]= pow((p1[i] - p2[i]), 2);
	for (int id = 0; id < n; id++) {
		*result += temp[id];
	}
	*result = sqrt(*result);
}
__global__ void manhattanDistance(dd *p1, dd *p2, dd *result, dd *temp) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	*result = 0;
	temp[i] = abs(p1[i] - p2[i]);
	for (int id = 0; id < n; id++) {
		*result += temp[id];
	}
}
__global__ void dotProduct(dd*a, dd*b, dd*c,dd *result) {  //for calculating a.b
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	c[i] = a[i] * b[i];
	*result = 0;
	for (int id = 0; id < n; id++) {
		*result += c[id];
	}
}
__global__ void euclidianDotProduct(dd*a, dd*b, dd*c,dd*result) { //for calculating ||a||.||b||
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	c[i] = (a[i] * a[i]);
	dd temp=0;
	for (int id = 0; id < n; id++) {
		temp += c[i];
	}
	*result = sqrt(temp);
	temp = 0;
	c[i] = (b[i] * b[i]);
	for (int id = 0; id < n; id++) {
		temp += c[i];
	}
	*result=sqrt(temp)*(*result);
}
int main()
{
	dd p1[n], p2[n],manhattan_distance=0,euclidian_distance=0,euclidian_dot=0,vector_dot=0;
	dd *dev_p1, *dev_p2,*dev_temp,*dev_result;
	hipMalloc((void**)&dev_p1, n * sizeof(dd)); //vector 1
	hipMalloc((void**)&dev_p2, n * sizeof(dd)); //vector 2
	hipMalloc((void**)&dev_temp, n * sizeof(dd));  
	hipMalloc(&dev_result, sizeof(dd));
	for (int i = 0; i < n; i++) {
		p1[i] = rand() / (double)RAND_MAX;
		p2[i] = rand() / (double)RAND_MAX;
	}
	cout << "\nVector 1: ";
	for (int i = 0; i < n; i++) {
		cout << setw(8) << setprecision(5) << p1[i] << " ";
		//cout << p1[i] << " ";
	}
	cout << "\n\nVector 2: ";
	for (int i = 0; i < n; i++) {
		cout <<setw(8)<<setprecision(5)<< p2[i] << " ";
		//cout << p2[i] << " ";
	}
	hipMemcpy(dev_p1, p1, n * sizeof(dd), hipMemcpyHostToDevice); //copy vectors into device
	hipMemcpy(dev_p2, p2, n * sizeof(dd), hipMemcpyHostToDevice);

	euclidianDistance << <1, n>> > (dev_p1, dev_p2, dev_result,dev_temp);
	hipMemcpy(&euclidian_distance, dev_result,sizeof(dd), hipMemcpyDeviceToHost); //copy euclidian distance from device to host
	cout <<"\n\nEuclidian Distance:"<< euclidian_distance<<endl; //print euclidian distance

	manhattanDistance << <1, n >> > (dev_p1, dev_p2, dev_result, dev_temp);
	hipMemcpy(&manhattan_distance, dev_result, sizeof(dd), hipMemcpyDeviceToHost); //copy manhattan distance from device to host
	cout << "\nManhattan Distance:" << manhattan_distance << endl; //print manhattan distance

	euclidianDotProduct << <1, n >> > (dev_p1, dev_p2,dev_temp, dev_result);
	hipMemcpy(&euclidian_dot, dev_result, sizeof(dd), hipMemcpyDeviceToHost);
	dotProduct << <1, n >> > (dev_p1, dev_p2, dev_temp, dev_result);
	hipMemcpy(&vector_dot, dev_result, sizeof(dd), hipMemcpyDeviceToHost);

	cout << "\nVector dot: " << vector_dot<<"\nEuclidian dot: "<<euclidian_dot;
	cout << "\nCosine Distance: " << (vector_dot / euclidian_dot)<<endl; //cosine distance = vector dot product / euclidian dot product
	
    return 0;
}