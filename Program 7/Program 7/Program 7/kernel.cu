#include "hip/hip_runtime.h"
/*
Design a parallel program in CUDA C++ platform for the following:
Find the transpose, sum, difference, scalar and vector multiplications of matrix of parallel
and randomly initialized with the number between -1.00 to +1.00. 
Mention the parameters: number of processors used, execution time and memory utilization
*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<math.h>
#include<iomanip>
#include<iostream>
#define dd double
#define n 4
using namespace std;
__global__ void matrixAdd(dd a[][n], dd b[][n], dd c[][n]) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	c[i][j] = a[i][j] + b[i][j];
}
__global__ void matrixSub(dd a[][n], dd b[][n], dd c[][n]) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	c[i][j] = a[i][j] - b[i][j];
}
__global__ void matrixTranspose(dd a[][n], dd b[][n]) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	b[i][j] = a[j][i];
}
__global__ void matrixMultiply(dd a[][n], dd b[][n], dd c[][n]) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	for (int k = 0; k < n; k++) {
		c[i][k] += a[i][k] * b[k][j];
	}
}
int main()
{
	dd a[n][n], b[n][n], c[n][n];
	dd(*dev_a)[n], (*dev_b)[n], (*dev_c)[n];
	hipMalloc((void**)&dev_a, n*n * sizeof(dd));
	hipMalloc((void**)&dev_b, n*n * sizeof(dd));
	hipMalloc((void**)&dev_c, n*n * sizeof(dd));
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i][j] = rand() / (dd)RAND_MAX;
			b[i][j] = rand() / (dd)RAND_MAX;
		}
	}
	hipMemcpy(dev_a, a, n*n * sizeof(dd), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n*n * sizeof(dd), hipMemcpyHostToDevice);
	dim3 block(n / 2, n / 2);  
	dim3 thread(n/(n/2), n/(n/2));
	matrixAdd << <block, thread >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, n*n * sizeof(dd), hipMemcpyDeviceToHost);
	cout<< "\nMatrix A: " << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << a[i][j];
		}
		cout << endl;
	}
	cout << "\nMatrix B: " << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << b[i][j];
		}
		cout << endl;
	}
	cout << "\nAddition Result: " << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << c[i][j];
		}
		cout << endl;
	}
	matrixSub << <block, thread >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, n*n * sizeof(dd), hipMemcpyDeviceToHost);
	cout << "\nSubtraction Result: " << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << c[i][j];
		}
		cout << endl;
	}
	matrixTranspose << <block, thread >> >(dev_a, dev_c);
	hipMemcpy(c, dev_c, n*n * sizeof(dd), hipMemcpyDeviceToHost);
	cout << "\nTranspose result of matrix A: " << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << c[i][j];
		}
		cout << endl;
	}
	dim3 mBlock(1, 1);
	dim3 mThread(n, n);
	matrixMultiply << <mBlock, mThread >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, n*n * sizeof(dd), hipMemcpyDeviceToHost);
	cout << "\nMultiplication AxB: \n" << endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << setw(8) << setprecision(4) << c[i][j];
		}
		cout << endl;
	}
    return 0;
}